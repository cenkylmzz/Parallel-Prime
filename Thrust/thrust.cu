#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>


using namespace std;


void calculateSQRTM(int number);
void calculateParallel();
void printPrimes(thrust::device_vector<int> parallelPrimes);


int M, PRIMES_SQRTM_SIZE, SQRT_M;
int k = 0;
int PARALLEL_PRIMES_SIZE = 0;
double SEQUENTIAL_TIME, PARALLEL_TIME;

int *PRIMES_SQRTM;  // calculated primes until SQRT(M)
int *d_PRIMES_SQRTM;


struct isPrimeFunctor{
    int *d_PRIMES_SQRTM;
    isPrimeFunctor(int* d_PRIMES_SQRTM) : d_PRIMES_SQRTM(d_PRIMES_SQRTM) {}

    __host__ __device__
    int operator()(int num) const{    // Using the SQRT(M) primes calculated sequentially, check the number is prime or not.
    int K, QUO, REM;
    K = 1;
    while (1){
        QUO = num / d_PRIMES_SQRTM[K];
        REM = num % d_PRIMES_SQRTM[K];
        if (REM == 0){
            return 0;
            break;
        }
        if (QUO <= d_PRIMES_SQRTM[K] || d_PRIMES_SQRTM[K+1] == 0){
            return num;
            break;
        }
        K += 1;
    }
    return 0;
    }
};

struct isNotZeroFunctor{
    __host__ __device__
    bool operator()(int num) const{
        return num != 0;
    }
};


int main(int argc, char** argv) {
	if (argc!=2){
		cout <<"Usage: ./<COMPILED_FILE_NAME> <M>\n"<<endl;
		return 0;
	}
    auto start = chrono::high_resolution_clock::now();

	M = atoi(argv[1]);
	SQRT_M = (int) sqrt(M);
    hipHostMalloc((void**)&PRIMES_SQRTM, M/2 * sizeof(int));
    calculateSQRTM(SQRT_M);

    hipMalloc((void**)&d_PRIMES_SQRTM, M /2 * sizeof(int));
    hipMemcpy(d_PRIMES_SQRTM, PRIMES_SQRTM, M/2 * sizeof(int), hipMemcpyHostToDevice);

    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed = end - start;
    SEQUENTIAL_TIME = elapsed.count();

    start = chrono::high_resolution_clock::now();
    calculateParallel();
    end = chrono::high_resolution_clock::now();
    elapsed = end - start;
    PARALLEL_TIME = elapsed.count();

    cout << SEQUENTIAL_TIME + PARALLEL_TIME << endl;

    return 0;
}



void calculateSQRTM(int number){                   // Calculate primes until sqrt(M) sequentially
    int K, J, QUO, REM, N;
    PRIMES_SQRTM_SIZE = 0;
    PRIMES_SQRTM[0] =  2;
    PRIMES_SQRTM[1] =  3;
    PRIMES_SQRTM_SIZE += 2;
    J = 1;
    for (N = 3; N <= number; N+=2){
        K = 1;
        while (1){
            QUO = N / PRIMES_SQRTM[K];
            REM = N % PRIMES_SQRTM[K];
            if (REM == 0){
                break;
            }
            if (QUO <= PRIMES_SQRTM[K] || PRIMES_SQRTM[K+1] == 0){
                PRIMES_SQRTM[J+1] = N;
                PRIMES_SQRTM_SIZE += 1;
                J += 1;
                break;
            }
            K += 1;
        }
    }
}
void calculateParallel(){

    int start_index = (SQRT_M % 2 == 0) ? (SQRT_M + 1) : SQRT_M + 2;
    int end_index = M;  
    int size = end_index % 2 == 0 ? (end_index - start_index) / 2 + 1 : (end_index - start_index) / 2 + 2;

    thrust::device_vector<int> device_numbers(size);
    thrust::device_vector<int> out_device_numbers(size);


    thrust::sequence(device_numbers.begin(), device_numbers.end(), start_index, 2);
    thrust::transform(device_numbers.begin(), device_numbers.end(), out_device_numbers.begin(), isPrimeFunctor(d_PRIMES_SQRTM));
    PARALLEL_PRIMES_SIZE = thrust::count_if(out_device_numbers.begin(), out_device_numbers.end(), isNotZeroFunctor());
    

    thrust::device_vector<int> parallelPrimes(PARALLEL_PRIMES_SIZE);
    thrust::copy_if(out_device_numbers.begin(), out_device_numbers.end(), parallelPrimes.begin(), isNotZeroFunctor());

    // printPrimes(parallelPrimes);         // Uncomment this line to print the primes calculated in parallel
}



void printPrimes(thrust::device_vector<int> parallelPrimes){
	vector<int> primes_vector = {};
	int i;
	for (i=0;i<PRIMES_SQRTM_SIZE;i++)
		primes_vector.push_back(PRIMES_SQRTM[i]);

	thrust::sort(parallelPrimes.begin(), parallelPrimes.end());

	for (i=0;i<parallelPrimes.size();i++){
        primes_vector.push_back(parallelPrimes[i]);
    }
	
	for (i=0;i<primes_vector.size();i++)
		cout << primes_vector[i] << " ";
	cout << endl;
}